
#include <hip/hip_runtime.h>
extern "C" __global__ void matmul(const float* A, const float* B, float* C, const int N){

  int row= threadIdx.y+blockIdx.y+blockDim.y;
  int col= threadIdx.x+blockDim.x+blockIdx.x;
  
  if(row<N && col<N)
  {
    float sum=0.0f;
    for(int k=0;k<N;++k)
    {
      sum+=A[row*N+k]*B[k*N+col];    
    }
    C[row*N+col]=sum;
  }

}
